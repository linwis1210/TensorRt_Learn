#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipDNN.h>
#include <cudnn_ops_infer.h>
using namespace std;

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      printf(hipdnnGetErrorString(status));                   \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

int main() {
  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);
  int in_channels = 1; //每层卷积的输入通道
  int out_channels = 9; //每层卷积的输出通道
  int batch_size = 1;
  int image_height = 1024; // 图片尺寸
  int image_width =  1024;

  int kernel_height = 3;//卷积核尺寸
  int kernel_width = 3;
  int padding = 1;
  int stride = 1 ;

  bool biasflag = true;

hipdnnTensorDescriptor_t input_descriptor;
checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW ,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/batch_size,
                                      /*channels=*/in_channels,
                                      /*image_height=*/image_height,
                                      /*image_width=*/image_width));

hipdnnTensorDescriptor_t output_descriptor;
checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/batch_size,
                                      /*channels=*/out_channels,
                                      /*image_height=*/image_height,
                                      /*image_width=*/image_width));

hipdnnTensorDescriptor_t z_descriptor;
checkCUDNN(hipdnnCreateTensorDescriptor(&z_descriptor));
checkCUDNN(hipdnnSetTensor4dDescriptor(z_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/batch_size,
                                      /*channels=*/out_channels,
                                      /*image_height=*/image_height,
                                      /*image_width=*/image_width));

hipdnnTensorDescriptor_t bias_descriptor;
checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
checkCUDNN(hipdnnSetTensor4dDescriptor(bias_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/batch_size,
                                      /*channels=*/out_channels,
                                      /*kernel_height=*/kernel_height,
                                      /*kernel_width=*/kernel_width));

hipdnnFilterDescriptor_t kernel_descriptor;
checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*out_channels=*/out_channels,
                                      /*in_channels=*/in_channels,
                                      /*kernel_height=*/kernel_height,
                                      /*kernel_width=*/kernel_width));

hipdnnConvolutionDescriptor_t convolution_descriptor;
checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                           /*pad_height=*/padding,
                                           /*pad_width=*/padding,
                                           /*vertical_stride=*/stride,
                                           /*horizontal_stride=*/stride,
                                           /*dilation_height=*/1,
                                           /*dilation_width=*/1,
                                           /*mode=*/HIPDNN_CROSS_CORRELATION,
                                           /*computeType=*/HIPDNN_DATA_FLOAT));

double coef = 100; //specifies the upper bound
hipdnnActivationDescriptor_t activationDesc;
checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
checkCUDNN(hipdnnSetActivationDescriptor(activationDesc, 
                                        /*hipdnnActivationMode_t*/ HIPDNN_ACTIVATION_RELU,
                                        /*hipdnnNanPropagation_t*/ HIPDNN_NOT_PROPAGATE_NAN,
                                        coef
                                        ));    

// hipdnnConvolutionFwdAlgoPerf_t convolution_a;
// checkCUDNN(
//     cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
//                                         input_descriptor,
//                                         kernel_descriptor,
//                                         convolution_descriptor,
//                                         output_descriptor,
//                                         HIPDNN_CONVOLUTION_FWD_ALGO_GEMM,
//                                         /*memoryLimitInBytes=*/0,
//                                         &convolution_a));

size_t workspace_bytes = 0;
hipdnnConvolutionFwdAlgo_t convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   convolution_descriptor,
                                                   output_descriptor,
                                                   convolution_algorithm,
                                                   &workspace_bytes));


//分配内存
void* d_workspace{nullptr};
hipMalloc(&d_workspace, workspace_bytes);

//batch_size * channels * height * width
int in_image_bytes = batch_size * in_channels * image_height * image_width * sizeof(float); //输入图片的尺寸
int out_image_bytes = batch_size * out_channels * image_height * image_width* sizeof(float); //输出图片的尺寸
int bias_bytes = batch_size * out_channels * kernel_height * kernel_width * sizeof(float);  //偏置值矩阵

float* d_input{nullptr};
hipMalloc(&d_input, in_image_bytes);

float* z_input{nullptr};
hipMalloc(&z_input, out_image_bytes);

float* bias_input{nullptr};
hipMalloc(&bias_input, bias_bytes);

float* d_output{nullptr};
hipMalloc(&d_output, out_image_bytes);
hipMemset(d_output, 0, out_image_bytes);

int h_kernel_bytes = out_channels * in_channels * kernel_height * kernel_width * sizeof(float); //卷积核参数尺寸

float* d_kernel{nullptr};
hipMalloc(&d_kernel, h_kernel_bytes);

//调用函数
const float alpha = 1, beta = 0;
if(biasflag == false){
checkCUDNN(hipdnnConvolutionForward(cudnn,
                                   &alpha,
                                   input_descriptor,
                                   d_input,
                                   kernel_descriptor,
                                   d_kernel,
                                   convolution_descriptor,
                                   convolution_algorithm,
                                   d_workspace,
                                   workspace_bytes,
                                   &beta,
                                   output_descriptor,
                                   d_output));
}else {
checkCUDNN(cudnnConvolutionBiasActivationForward(cudnn,
                                   &alpha,
                                   input_descriptor,
                                   d_input,
                                   kernel_descriptor,
                                   d_kernel,
                                   convolution_descriptor,
                                   convolution_algorithm,
                                   d_workspace,
                                   workspace_bytes,
                                   &beta,
                                   z_descriptor,
                                   z_input,
                                   bias_descriptor,
                                   bias_input,
                                   activationDesc,
                                   output_descriptor,
                                   d_output));
}
float* h_output = new float[out_image_bytes];
hipMemcpy(h_output, d_output, out_image_bytes, hipMemcpyDeviceToHost);

// Do something with h_output ...

//释放内存
delete[] h_output;
hipFree(d_kernel);
hipFree(d_input);
hipFree(d_output);
hipFree(d_workspace);
hipFree(z_input);
hipFree(bias_input);

hipdnnDestroyTensorDescriptor(input_descriptor);
hipdnnDestroyTensorDescriptor(output_descriptor);
hipdnnDestroyFilterDescriptor(kernel_descriptor);
hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
hipdnnDestroyTensorDescriptor(z_descriptor);
hipdnnDestroyTensorDescriptor(bias_descriptor);
hipdnnDestroyActivationDescriptor(activationDesc);
hipdnnDestroy(cudnn);
};